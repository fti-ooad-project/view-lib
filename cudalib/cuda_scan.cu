#include "hip/hip_runtime.h"
#include <cudalib\cuda_scan.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cudalib\cudaUtil.h>
//int *c_scan_buffer;

CudaScanHandler *CudaScanHandler::getSingleton()
{
	static CudaScanHandler *sngl = new CudaScanHandler();
	return sngl;
}
void CudaScanHandler::init( uint n )
{
	if( isInited() )
		return;
	setInited( true );
	hipError_t err = hipSuccess;
	//checkError( hipMalloc( &c_scan_buffer , n * sizeof( int ) ) , "cuda scan handler malloc buffer" );
error:
	return;
}
int CudaScanHandler::scanAdd( int *obuf , int *ibuf , uint n ) const
{
	dim3 thread_count( min( n , 32 ) , 1 );
	dim3 block_count( ( n + thread_count.x - 1 ) / thread_count.x , 1 );
	c_scan << < block_count , thread_count >> >( obuf , ibuf , n );
	hipError_t err = hipSuccess;
	checkError( hipGetLastError() , "cuda scan handler scanAdd()" );
error:
	return;
}
void CudaScanHandler::release()
{
	if( !isInited() )
		return;
	setInited( false );
	hipError_t err = hipSuccess;
	//checkError( hipFree( c_scan_buffer ) , "cuda scan handler free buffer" );
error:
	return;
}