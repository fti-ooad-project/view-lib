#include "hip/hip_runtime.h"
#include <cudalinalg/vec.h>
#include<hiprand/hiprand_kernel.h>
DEVICE float PI;
//#define RAND_COUNT 0x1000
#define RANDSTATE_COUNT 0x100
//__device__ float randf[ RAND_COUNT ];
DEVICE hiprandState randstate[ RANDSTATE_COUNT ];
//__device__ unsigned int cur_rand;
DEVICE float c_wrap( float x , float min , float max )
{
	if( x > max ) return max;
	if( x < min ) return min;
	return x;
}
DEVICE float c_sqr( float i )
{
	return i * i;
}
GLOBAL void devInit()
{
	PI = acosf( -1.0f );
}
GLOBAL void randInit( unsigned int n )
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if( i < n )
	{
		hiprand_init( 93735373UL + 56733797UL * i , i , 0 , randstate + i );
	}
}
LOCAL void VectorFactory::init()
{
	devInit << < 1 , 1 >> >();
	unsigned int thread_count = min( RANDSTATE_COUNT , 1024 );
	unsigned int block_count = ( RANDSTATE_COUNT + thread_count - 1 ) / thread_count;
	randInit << < block_count , thread_count >> >( RANDSTATE_COUNT );
}
DEVICE float VectorFactory::getRandom( unsigned int seed )
{
	return hiprand_uniform( randstate + ( seed % RANDSTATE_COUNT ) );
}
DEVICE float VectorFactory::getPI()
{
	return PI;
}
DEVICE f3 VectorFactory::getRandomHalfSphere( unsigned int seed )
{
	float phi = getRandom( seed ) * PI * 2.0f;
	float cp = cosf( phi );
	float sp = sinf( phi );
	float ct = getRandom( seed );
	float st = sqrtf( 1.0f - ct * ct );
	return f3( st * cp , st * sp , ct );
}
DEVICE f3 VectorFactory::getRandomSphere( unsigned int seed )
{
	float phi = getRandom( seed ) * PI * 2.0f;
	float cp = cosf( phi );
	float sp = sinf( phi );
	float ct = -1.0f + 2.0f * getRandom( seed );
	float st = sqrtf( 1.0f - ct * ct );
	return f3( st * cp , st * sp , ct );
}
DEVICE f2 VectorFactory::getRandomCircle( unsigned int seed )
{
	float phi = getRandom( seed ) * PI * 2.0f;
	float r = powf( getRandom( seed ) , 0.5f );
	return f2( cos( phi ) , sin( phi ) ) * r;
}
DEVICE f3 VectorFactory::getReflected( f3 const &v , f3 const &n )
{
	return v - 2.0f * n * ( n * v );
}
DEVICE f3 VectorFactory::getRefracted( f3 const &v , f3 const &n , float kn )
{
	float cosa = -v * n;
	float sina2 = 1.0f - cosa * cosa;
	float sinb2 = sina2 * kn * kn;
	if( sinb2 > 1.0f )
		return getReflected( v , n );
	float cosb = sqrtf( 1.0f - sinb2 );
	return n * ( -cosb + cosa * kn ) + v * kn;
}
DEVICE f3 VectorFactory::getDiffuseReflected( f3 const &v , f3 const &n , float spec , unsigned int seed )
{
	f3 locx = vecx( v , n ).g_norm();
	f3 locy = vecx( locx , n );
	f3 kvec = getRandomHalfSphere( seed );
	f3 rand_vec = n * kvec.z() + locx * kvec.x() + locy * kvec.y();
	f3 refl = getReflected( v , n );
	f3 out = ( refl * spec + rand_vec * ( 1.0 - spec ) ).g_norm();
	return out;
}
DEVICE f3 VectorFactory::getDiffuseRefracted( f3 const &v , f3 const &n , float spec , float kn , unsigned int seed )
{
	f3 locx = vecx( v , n ).g_norm();
	f3 locy = vecx( locx , n );
	f3 kvec = getRandomHalfSphere( seed );
	f3 rand_vec = n * kvec.z() + locx * kvec.x() + locy * kvec.y();
	f3 refl = getRefracted( v , n , kn );
	f3 out = ( refl * spec + rand_vec * ( 1.0 - spec ) ).g_norm();
	return out;
}