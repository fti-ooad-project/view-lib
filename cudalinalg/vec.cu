#include "hip/hip_runtime.h"
#include "linalg/vec.h"
#include<hiprand/hiprand_kernel.h>
DEVICE float PI;
//#define RAND_COUNT 0x1000
#define RANDSTATE_COUNT 0x100
//__device__ float randf[ RAND_COUNT ];
DEVICE hiprandState randstate[ RANDSTATE_COUNT ];
//__device__ unsigned int cur_rand;
DEVICE float wrap( float x , float min , float max )
{
	if( x > max ) return max;
	if( x < min ) return min;
	return x;
}
DEVICE float sqr( float i )
{
	return i * i;
}
GLOBAL void devInit()
{
	PI = acosf( -1.0f );
}
GLOBAL void randInit( unsigned int n )
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if( i < n )
	{
		hiprand_init( 93735373UL + 56733797UL * i , i , 0 , randstate + i );
	}
}
LOCAL void VectorFactory::init()
{
	devInit << < 1 , 1 >> >();
	unsigned int thread_count = min( RANDSTATE_COUNT , 1024 );
	unsigned int block_count = ( RANDSTATE_COUNT + thread_count - 1 ) / thread_count;
	randInit << < block_count , thread_count >> >( RANDSTATE_COUNT );
}
DEVICE float VectorFactory::getRandom( unsigned int seed )
{
	return hiprand_uniform( randstate + ( seed % RANDSTATE_COUNT ) );
}
DEVICE float VectorFactory::getPI()
{
	return PI;
}
DEVICE f3 VectorFactory::getRandomHalfSphere( unsigned int seed )
{
	float phi = getRandom( seed ) * PI * 2.0f;
	float cp = cosf( phi );
	float sp = sinf( phi );
	float ct = getRandom( seed );
	float st = sqrtf( 1.0f - ct * ct );
	return f3( st * cp , st * sp , ct );
}
DEVICE f3 VectorFactory::getRandomSphere( unsigned int seed )
{
	float phi = getRandom( seed ) * PI * 2.0f;
	//float theta = PI * 0.5f + asinf( 2.0f * getRandom() - 1.0f );
	float cp = cosf( phi );
	float sp = sinf( phi );
	float ct = -1.0f + 2.0f * getRandom( seed );
		//cosf( theta );
	float st = sqrtf( 1.0f - ct * ct );
		//sinf( theta );
	return f3( st * cp , st * sp , ct );
}
DEVICE f2 VectorFactory::getRandomCircle( unsigned int seed )
{
	float phi = getRandom( seed ) * PI * 2.0f;
	float r = powf( getRandom( seed ) , 0.5f );
	return f2( cos( phi ) , sin( phi ) ) * r;
}
DEVICE f3 VectorFactory::getReflected( f3 const &v , f3 const &n )
{
	return v - 2.0f * n * ( n * v );
}