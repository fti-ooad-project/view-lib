#include "hip/hip_runtime.h"
#include <cudalinalg/VectorFactory.h>
#include<hiprand/hiprand_kernel.h>
DEVICE float PI;
//#define RAND_COUNT 0x1000
#define RANDSTATE_COUNT 0x100
//__device__ float randf[ RAND_COUNT ];
DEVICE hiprandState randstate[ RANDSTATE_COUNT ];
//__device__ unsigned int cur_rand;
GLOBAL void devInit()
{
	PI = acosf( -1.0f );
}
GLOBAL void randInit( unsigned int n )
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if( i < n )
	{
		hiprand_init( 93735373UL + 56733797UL * i , i , 0 , randstate + i );
	}
}
LOCAL void VectorFactory::init()
{
	devInit << < 1 , 1 >> >();
	unsigned int thread_count = min( RANDSTATE_COUNT , 1024 );
	unsigned int block_count = ( RANDSTATE_COUNT + thread_count - 1 ) / thread_count;
	randInit << < block_count , thread_count >> >( RANDSTATE_COUNT );
}
DEVICE float VectorFactory::getRandom( unsigned int seed )
{
	return hiprand_uniform( randstate + ( seed % RANDSTATE_COUNT ) );
}
DEVICE float VectorFactory::getPI()
{
	return PI;
}
DEVICE df3 VectorFactory::getRandomHalfSphere( unsigned int seed )
{
	float phi = getRandom( seed ) * PI * 2.0f;
	float cp = cosf( phi );
	float sp = sinf( phi );
	float ct = getRandom( seed );
	float st = sqrtf( 1.0f - ct * ct );
	return df3( st * cp , st * sp , ct );
}
DEVICE df3 VectorFactory::getRandomCosHalfSphere( unsigned int seed )
{
	float phi = getRandom( seed ) * PI * 2.0f;
	float cp = cosf( phi );
	float sp = sinf( phi );
	float ct = cosf( getRandom( seed ) * PI * 0.5f );
	float st = sqrtf( 1.0f - ct * ct );
	return df3( st * cp , st * sp , ct );
}
DEVICE df3 VectorFactory::getRandomSphere( unsigned int seed )
{
	float phi = getRandom( seed ) * PI * 2.0f;
	float cp = cosf( phi );
	float sp = sinf( phi );
	float ct = -1.0f + 2.0f * getRandom( seed );
	float st = sqrtf( 1.0f - ct * ct );
	return df3( st * cp , st * sp , ct );
}
DEVICE df2 VectorFactory::getRandomCircle( unsigned int seed )
{
	float phi = getRandom( seed ) * PI * 2.0f;
	float r = powf( getRandom( seed ) , 0.5f );
	return df2( cos( phi ) , sin( phi ) ) * r;
}
DEVICE df3 VectorFactory::getReflected( df3 const &v , df3 const &n )
{
	return v - 2.0f * n * ( n * v );
}
DEVICE df3 VectorFactory::getRefracted( df3 const &v , df3 const &n , float kn )
{
	float cosa = -v * n;
	float sina2 = 1.0f - cosa * cosa;
	float sinb2 = sina2 * kn * kn;
	if( sinb2 > 1.0f )
		return getReflected( v , n );
	float cosb = sqrtf( 1.0f - sinb2 );
	return n * ( -cosb + cosa * kn ) + v * kn;
}
DEVICE df3 VectorFactory::getDiffuseReflected( df3 const &v , df3 const &n , float spec , unsigned int seed )
{
	df3 locx = vecx( v , n ).g_norm();
	df3 locy = vecx( locx , n );
	df3 kvec = getRandomCosHalfSphere( seed );
	df3 rand_vec = n * kvec.z() + locx * kvec.x() + locy * kvec.y();
	df3 refl = getReflected( v , n );
	df3 out = ( refl * spec + rand_vec * ( 1.0 - spec ) ).g_norm();
	return out;
}
DEVICE df3 VectorFactory::getDiffuseRefracted( df3 const &v , df3 const &n , float spec , float kn , unsigned int seed )
{
	df3 locx = vecx( v , n ).g_norm();
	df3 locy = vecx( locx , n );
	df3 kvec = getRandomCosHalfSphere( seed );
	df3 rand_vec = n * kvec.z() + locx * kvec.x() + locy * kvec.y();
	df3 refl = getRefracted( v , n , kn );
	df3 out = ( refl * spec + rand_vec * ( 1.0 - spec ) ).g_norm();
	return out;
}