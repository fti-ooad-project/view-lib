#include "hip/hip_runtime.h"
#include <cudalinalg\cudaMatrixSolver.h>
#include ""
#include <cudalinalg\vec.h>
#define getValue( i , j ) mat[ i * mat_size + j ]
__global__ void dev_triangulate( float *mat , int mat_size , int n )
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	if( i < mat_size && j < mat_size && i > n )
	{
		float k = getValue( n , n );
		if( fabsf( k ) > 0.0f )
		{
			getValue( i , j ) -= getValue( n , j ) * getValue( i , n ) / k;
		} else
		{
			//getValue( i , j ) = 0.0f;
		}
	}
}
__global__ void dev_calcNullRow( float *mat , int mat_size , int *outv )
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if( i < mat_size )
	{
		float sum = 0.0f;
		for( int k = 0; k < mat_size; k++ )
		{
			float v = getValue( i , k );
			sum += v * v;
		}
		if( sum < 0.000001f )
			*outv = i;
	}
}
__global__ void dev_shiftY( float *out_mat , float *mat , int mat_size , int zerorow )
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	if( i < mat_size && j < mat_size )
	{
		int shift = 0;
		if( i >= zerorow )
			shift = 1;
		if( i == mat_size - 1 )
		{
			out_mat[ i * mat_size + j ] = 0.0f;
		} else
		{
			out_mat[ i * mat_size + j ] = mat[ ( i + shift ) * mat_size + j ];
		}
	}
}
CudaMatrixSolver *CudaMatrixSolver::getSingleton()
{
	static cudaMatrixSolver *sngl = new cudaMatrixSolver();
	return sngl;
}
void CudaMatrixSolver::makeSpace( int size )
{
	free( _host_buffer );
	hipFree( _dev_buffer );
	_buffer_size = size;
	_host_buffer = malloc( _buffer_size );
	hipMalloc( &_dev_buffer , _buffer_size );
}
/*take some memory on host and device*/
void CudaMatrixSolver::init()
{
	if( isInited() ) return;
	setInited( true );
	_buffer_size = 0x1000;
	_host_buffer = malloc( _buffer_size );
	hipMalloc( &_dev_buffer , _buffer_size );
}
/*return true if kernel value is match and false if not*/
bool CudaMatrixSolver::calcKernelVector( float const * in_matrix , float in_kernel_value , int matrix_size , float * out_kernel_vector )
{
	int byte_size = matrix_size * matrix_size * sizeof( float );
	if( 2 * byte_size > _buffer_size )
	{
		makeSpace( byte_size * 2 );
	}
	float *dev_mat = ( float* )_dev_buffer;
	hipMemcpy( dev_mat , in_matrix , byte_size , hipMemcpyHostToDevice );
	dim3 block_size = dim3( matrix_size / 32 + 1 , matrix_size / 32 + 1 );
	for( int n = 0; n < matrix_size; n++ )
	{
		dev_triangulate<<< block_size , dim3( 32 , 32 ) >>>( dev_mat , matrix_size , n );
		hipDeviceSynchronize();
	}
	int zerorow = -1;
	{
		int *dev_zerorow;
		hipMalloc( &dev_zerorow , sizeof( int ) );
		hipMemcpy( dev_zerorow , &zerorow , sizeof( int ) , hipMemcpyHostToDevice );
		dev_calcNullRow << < 1 , 32 >> >( dev_mat , matrix_size , dev_zerorow );
		hipDeviceSynchronize();
		hipMemcpy( &zerorow , dev_zerorow , sizeof( int ) , hipMemcpyDeviceToHost );
		hipFree( dev_zerorow );
	}
	if( zerorow < 0 )
	{
		return false;
	}
	float *dev_temp_mat = ( float* )_dev_buffer + matrix_size * matrix_size;
	dev_shiftY << < block_size , dim3( 32 , 32 ) >> >( dev_temp_mat , dev_mat , matrix_size , zerorow );
	hipMemcpy( _host_buffer , dev_temp_mat , byte_size , hipMemcpyDeviceToHost );
	float *local_mat = ( float* )_host_buffer;
	out_kernel_vector[ matrix_size - 1 ] = 1.0f;
	for( int i = matrix_size - 2; i >= 0; i-- )
	{
		out_kernel_vector[ i ] = 0.0f;
		for( int j = i + 1; j < matrix_size; j++ )
		{
			out_kernel_vector[ i ] -= out_kernel_vector[ j ] * local_mat[ i * matrix_size + j ];
		}
		out_kernel_vector[ i ] /= local_mat[ i * matrix_size + i ];
	}
	return true;
}
/*free memory on host and device*/
void CudaMatrixSolver::release()
{
	if( !isInited() ) return;
	setInited( false );
}