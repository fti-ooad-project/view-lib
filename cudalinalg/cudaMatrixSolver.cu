#include "hip/hip_runtime.h"
#include <cudalinalg\cudaMatrixSolver.h>
#include ""
#include <cudalinalg\vec.h>
#include <iomanip>
#define getValue( i , j ) mat[ i * mat_size + j ]
__global__ void dev_triangulate( float *mat , int mat_size , int n )
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	if( i < mat_size && j < mat_size && i > n )
	{
		float k = getValue( n , n );
		if( fabsf( k ) > 0.0001f )
		{
			getValue( i , j ) -= getValue( n , j ) * getValue( i , n ) / k;
		} else
		{
			//getValue( i , j ) = 0.0f;
		}
	}
}
__global__ void dev_calcNullRow( float *mat , int mat_size , int *outv )
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if( i < mat_size )
	{
		float sum = 0.0f;
		for( int k = 0; k < mat_size; k++ )
		{
			float v = getValue( i , k );
			sum += v * v;
		}
		if( sum < 0.01f )
			*outv = i;
	}
}
__global__ void dev_shiftY( float *out_mat , float *mat , int mat_size , int zerorow )
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	if( i < mat_size && j < mat_size )
	{
		int shift = 0;
		if( i >= zerorow )
			shift = 1;
		if( i == mat_size - 1 )
		{
			out_mat[ i * mat_size + j ] = 0.0f;
		} else
		{
			out_mat[ i * mat_size + j ] = mat[ ( i + shift ) * mat_size + j ];
		}
	}
}
__global__ void dev_subKernelValue( float *mat , int mat_size , float value )
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	if( j < mat_size )
	{
		mat[ j * mat_size + j ] -= value;
	}
}
CudaMatrixSolver *CudaMatrixSolver::getSingleton()
{
	static CudaMatrixSolver *sngl = new CudaMatrixSolver();
	return sngl;
}
void CudaMatrixSolver::makeSpace( int size )
{
	free( _host_buffer );
	hipFree( _dev_buffer );
	_buffer_size = size;
	_host_buffer = malloc( _buffer_size );
	hipMalloc( &_dev_buffer , _buffer_size );
}
/*take some memory on host and device*/
void CudaMatrixSolver::init()
{
	if( isInited() ) return;
	setInited( true );
	_buffer_size = 0x1000;
	_host_buffer = malloc( _buffer_size );
	hipMalloc( &_dev_buffer , _buffer_size );
}
#define printMatrix( mat , N )\
{\
	hipMemcpy( _host_buffer , mat , N * N * sizeof( float ) , hipMemcpyDeviceToHost );\
	float *local_mat = ( float* )_host_buffer;\
	for( int ti = 0; ti < N; ti ++ )\
		{\
		for( int tj = 0; tj < N; tj++ )\
				{\
					std::cout << std::setprecision( 1 ) << std::setw( 4 ) << local_mat[ ti * N + tj ] << " ";\
				}\
		std::cout << "\n";\
	}\
}
/*return matrix determinant*/
void CudaMatrixSolver::calcKernelVector( float const * in_matrix , float kernel_value , int matrix_size , float * out_kernel_vector )
{
	int byte_size = matrix_size * matrix_size * sizeof( float );
	if( 2 * byte_size > _buffer_size )
	{
		makeSpace( byte_size * 2 );
	}
	float *dev_mat = ( float* )_dev_buffer;
	hipMemcpy( dev_mat , in_matrix , byte_size , hipMemcpyHostToDevice );
	dim3 block_size = dim3( matrix_size / 32 + 1 , matrix_size / 32 + 1 );
	dev_subKernelValue<<< block_size.x , 32 >>>( dev_mat , matrix_size , kernel_value );
	hipDeviceSynchronize();
	float *local_mat = ( float* )_host_buffer;
	/*hipMemcpy( local_mat , dev_mat , byte_size , hipMemcpyDeviceToHost );
	float det = calcMatrixDet2M( local_mat , matrix_size , 0.0f );
	if( fabsf( det ) > 0.0001f )
	{
		std::cout << "calcKernelVector: matrix determinant is not 0\n";
		return;
	}*/
	for( int n = 0; n < matrix_size; n++ )
	{
		dev_triangulate<<< block_size , dim3( 32 , 32 ) >>>( dev_mat , matrix_size , n );
		hipDeviceSynchronize();

	}
	int zerorow = -1;
	{
		int *dev_zerorow;
		hipMalloc( &dev_zerorow , sizeof( int ) );
		hipMemcpy( dev_zerorow , &zerorow , sizeof( int ) , hipMemcpyHostToDevice );
		dev_calcNullRow << < 1 , 32 >> >( dev_mat , matrix_size , dev_zerorow );
		hipDeviceSynchronize();
		hipMemcpy( &zerorow , dev_zerorow , sizeof( int ) , hipMemcpyDeviceToHost );
		hipFree( dev_zerorow );
	}
	if( zerorow < 0 )
	{
		//std::cout << "calcKernelVector: triangulation error\n";
		return;
	}
	//printMatrix( dev_mat , matrix_size );
	std::cout << kernel_value << "\n";
	float *dev_temp_mat = ( float* )_dev_buffer + matrix_size * matrix_size;
	dev_shiftY << < block_size , dim3( 32 , 32 ) >> >( dev_temp_mat , dev_mat , matrix_size , zerorow );
	hipMemcpy( _host_buffer , dev_temp_mat , byte_size , hipMemcpyDeviceToHost );
	out_kernel_vector[ matrix_size - 1 ] = 1.0f;
	for( int i = matrix_size - 2; i >= 0; i-- )
	{
		out_kernel_vector[ i ] = 0.0f;
		if( fabsf( local_mat[ i * matrix_size + i ] ) > 0.001f )
		{
			for( int j = i + 1; j < matrix_size; j++ )
			{
				out_kernel_vector[ i ] -= out_kernel_vector[ j ] * local_mat[ i * matrix_size + j ];
			}
			out_kernel_vector[ i ] /= local_mat[ i * matrix_size + i ];
		} else
		{
			out_kernel_vector[ i ] = 1.0f;
		}
	}
	return;
}
__global__ void dev_calcCellDeterm2M( float const *in_matrix , float *out_matrix , int cell_count )
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	if( i < cell_count && j < cell_count )
	{
		int indx = i * cell_count * 4 + j * 2;
		out_matrix[ i * cell_count + j ] = in_matrix[ indx ] * in_matrix[ indx + cell_count * 2 + 1 ] - in_matrix[ indx + cell_count * 2 ] * in_matrix[ indx + 1 ];
	}
}
float CudaMatrixSolver::calcMatrixDet2M( float const *in_matrix , int matrix_size , float kernel_value )
{
	for( int i = 1 << 30; i > 1; i >>= 1 )
	{
		if( i & matrix_size )
		{
			if( i != matrix_size )
			{
				std::cout << "calcMatrixDet2M: matrix size is not power of two\n";
				return 0.0f;
			} else
			{
				break;
			}
		}
	}
	int byte_size = matrix_size * matrix_size * sizeof( float );
	if( 2 * byte_size > _buffer_size )
	{
		makeSpace( byte_size * 2 );
	}
	float *dev_mat[ 2 ] = { ( float* )_dev_buffer , ( float* )_dev_buffer + matrix_size * matrix_size };
	hipMemcpy( dev_mat[ 0 ] , in_matrix , byte_size , hipMemcpyHostToDevice );
	if( kernel_value != 0.0f )
	{
		dim3 block_size = dim3( matrix_size / 32 + 1 , matrix_size / 32 + 1 );
		dev_subKernelValue << < block_size.x , 32 >> >( dev_mat[ 0 ] , matrix_size , kernel_value );
		hipDeviceSynchronize();
	}
	int cur = 1 , last = 0;
	//printMatrix( dev_mat[ 0 ] , 4 );
	for( int i = matrix_size / 2; i > 0; i >>= 1 )
	{
		dim3 block_size = dim3( i / 32 + 1 , i / 32 + 1 );
		dev_calcCellDeterm2M <<< block_size , dim3( 32 , 32 ) >> > ( dev_mat[ last ] , dev_mat[ cur ] , i );
		hipDeviceSynchronize();
		//printMatrix( dev_mat[ cur ] , i );
		last = cur;
		cur = 1 - cur;
	}
	float det;
	hipMemcpy( &det , dev_mat[ last ] , sizeof( float ) , hipMemcpyDeviceToHost );
	return det;
}
/*free memory on host and device*/
void CudaMatrixSolver::release()
{
	if( !isInited() ) return;
	setInited( false );
	free( _host_buffer );
	hipFree( _dev_buffer );
}