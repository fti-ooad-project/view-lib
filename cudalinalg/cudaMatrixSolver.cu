#include "hip/hip_runtime.h"
#include <cudalinalg\cudaMatrixSolver.h>
#include ""
#include <cudalinalg\vec.h>
#include <iomanip>
#define getValue( i , j ) mat[ i * mat_size + j ]
__global__ void dev_triangulate( float *mat , int mat_size , int n )
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	if( i < mat_size && j < mat_size && i > n )
	{
		float k = getValue( n , n );
		if( fabsf( k ) > 0.0f )
		{
			getValue( i , j ) -= getValue( n , j ) * getValue( i , n ) / k;
		} else
		{
			//getValue( i , j ) = 0.0f;
		}
	}
}
__global__ void dev_calcNullRow( float *mat , int mat_size , int *outv )
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if( i < mat_size )
	{
		float sum = 0.0f;
		for( int k = 0; k < mat_size; k++ )
		{
			float v = getValue( i , k );
			sum += v * v;
		}
		if( sum < 0.0001f )
			*outv = i;
	}
}
__global__ void dev_shiftY( float *out_mat , float *mat , int mat_size , int zerorow )
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	if( i < mat_size && j < mat_size )
	{
		int shift = 0;
		if( i >= zerorow )
			shift = 1;
		if( i == mat_size - 1 )
		{
			out_mat[ i * mat_size + j ] = 0.0f;
		} else
		{
			out_mat[ i * mat_size + j ] = mat[ ( i + shift ) * mat_size + j ];
		}
	}
}
__global__ void dev_subKernelValue( float *mat , int mat_size , float value )
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	if( j < mat_size )
	{
		mat[ j * mat_size + j ] -= value;
	}
}
CudaMatrixSolver *CudaMatrixSolver::getSingleton()
{
	static CudaMatrixSolver *sngl = new CudaMatrixSolver();
	return sngl;
}
void CudaMatrixSolver::makeSpace( int size )
{
	free( _host_buffer );
	hipFree( _dev_buffer );
	_buffer_size = size;
	_host_buffer = malloc( _buffer_size );
	hipMalloc( &_dev_buffer , _buffer_size );
}
/*take some memory on host and device*/
void CudaMatrixSolver::init()
{
	if( isInited() ) return;
	setInited( true );
	_buffer_size = 0x1000;
	_host_buffer = malloc( _buffer_size );
	hipMalloc( &_dev_buffer , _buffer_size );
}
#define printMatrix( mat )\
{\
	hipMemcpy( _host_buffer , mat , byte_size , hipMemcpyDeviceToHost );\
	float *local_mat = ( float* )_host_buffer;\
	for( int i = 0; i < matrix_size; i ++ )\
		{\
		for( int j = 0; j < matrix_size; j++ )\
				{\
			std::cout << std::setw( 2 ) << local_mat[ i * matrix_size + j ] << " ";\
				}\
		std::cout << "\n";\
	}\
}
/*return true if kernel value is match and false if not*/
bool CudaMatrixSolver::calcKernelVector( float const * in_matrix , float kernel_value , int matrix_size , float * out_kernel_vector )
{
	int byte_size = matrix_size * matrix_size * sizeof( float );
	if( 2 * byte_size > _buffer_size )
	{
		makeSpace( byte_size * 2 );
	}
	float *dev_mat = ( float* )_dev_buffer;
	hipMemcpy( dev_mat , in_matrix , byte_size , hipMemcpyHostToDevice );
	dim3 block_size = dim3( matrix_size / 32 + 1 , matrix_size / 32 + 1 );
	dev_subKernelValue<<< block_size.x , 32 >>>( dev_mat , matrix_size , kernel_value );
	for( int n = 0; n < matrix_size; n++ )
	{
		dev_triangulate<<< block_size , dim3( 32 , 32 ) >>>( dev_mat , matrix_size , n );
		hipDeviceSynchronize();
	}
	int zerorow = -1;
	{
		int *dev_zerorow;
		hipMalloc( &dev_zerorow , sizeof( int ) );
		hipMemcpy( dev_zerorow , &zerorow , sizeof( int ) , hipMemcpyHostToDevice );
		dev_calcNullRow << < 1 , 32 >> >( dev_mat , matrix_size , dev_zerorow );
		hipDeviceSynchronize();
		hipMemcpy( &zerorow , dev_zerorow , sizeof( int ) , hipMemcpyDeviceToHost );
		hipFree( dev_zerorow );
	}
	printMatrix( dev_mat );
	if( zerorow < 0 )
	{
		return false;
	}
	float *dev_temp_mat = ( float* )_dev_buffer + matrix_size * matrix_size;
	dev_shiftY << < block_size , dim3( 32 , 32 ) >> >( dev_temp_mat , dev_mat , matrix_size , zerorow );
	hipMemcpy( _host_buffer , dev_temp_mat , byte_size , hipMemcpyDeviceToHost );
	float *local_mat = ( float* )_host_buffer;
	out_kernel_vector[ matrix_size - 1 ] = 1.0f;
	for( int i = matrix_size - 2; i >= 0; i-- )
	{
		out_kernel_vector[ i ] = 0.0f;
		for( int j = i + 1; j < matrix_size; j++ )
		{
			out_kernel_vector[ i ] -= out_kernel_vector[ j ] * local_mat[ i * matrix_size + j ];
		}
		out_kernel_vector[ i ] /= local_mat[ i * matrix_size + i ];
	}
	return true;
}
/*free memory on host and device*/
void CudaMatrixSolver::release()
{
	if( !isInited() ) return;
	setInited( false );
	free( _host_buffer );
	hipFree( _dev_buffer );
}