#include "hip/hip_runtime.h"
#include <MoleculeSimulator\cudaMoleculeUtils.h>
#include <cudalinalg\vec.h>
#include ""
#include <iostream>
struct SortCube
{
	static const int N = 32;
	static const int DIV = 6;
	static const int SIZE = 60;
	int _array[ DIV ][ DIV ][ DIV ][ N + 1 ];
};
__device__ df3 calcForce( df3 const &p1 , df3 const &p2 )
{
	df3 dr = p2 - p1;
	const float l0 = 2.0f;
	const float u0 = 15.0f;
	float l = dr.g_mod();
	float ql;
	ql = powf( l0 / l , 2.0f );
	return dr * u0 * ( 6.0f * powf( ql , 4.0f ) - 12.0f * powf( ql , 7.0f ) );
}
__device__ __host__ void getIndex( int *outi3 , float const *inf3 )
{
	ito( 3 )
	{
		outi3[ i ] = ( inf3[ i ] + SortCube::SIZE * 0.5f ) / SortCube::SIZE * SortCube::DIV;
		outi3[ i ] = outi3[ i ] > 0 ? outi3[ i ] : 0;
		outi3[ i ] = outi3[ i ] < SortCube::DIV ? outi3[ i ] : SortCube::DIV - 1;
	}
}
__global__ void dev_calcLennardForceMatrix( void const *pos , void *force_mat , int count )
{
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	/*const int SHN = 64;
	__shared__ df3 shared_pos[ SHN ];
	if( threadIdx.x == 0 )
	{
		shared_pos[ threadIdx.y ] = ( ( df3* )pos )[ tidx ];
	}
	if( threadIdx.y == 0 )
	{
		shared_pos[ 32 + threadIdx.x ] = ( ( df3* )pos )[ tidy ];
	}
	__syncthreads();*/
	if( tidx > tidy && tidy < count && tidx < count )
	{
		df3 curpos = ( ( df3* )pos )[ tidy ];// shared_pos[ 32 + threadIdx.x ];
		df3 iterpos = ( ( df3* )pos )[ tidx ];// shared_pos[ threadIdx.y ];
		df3 dr = iterpos - curpos;
		df3 force( 0.0f );
		if( fabsf( dr.x() ) < 20.0f && fabsf( dr.y() ) < 20.0f )
		{
			const float l0 = 4.0f;
			const float u0 = 15.0f;
			float l = dr.g_mod();
			float ql;
			if( l > l0 / 2.0f )
			{
				ql = powf( l0 / l , 2.0f );
			}
			else
			{
				ql = powf( 2.0f , 2.0f );
			}
			force = dr * u0 * ( 6.0f * powf( ql , 4.0f ) - 12.0f * powf( ql , 7.0f ) );
		}
		( ( df3* )force_mat )[ tidy * count + tidx ] = force;
		( ( df3* )force_mat )[ tidx * count + tidy ] = -force;
	}
}
__global__ void dev_calcLennardForceVector( void const *force_mat , void *force_vec , int count )
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if( tid < count )
	{
		df3 temp_force( 0.0f );
		for( int j = 0; j < count; ++j )
		{
			if( j != tid )
			{
				temp_force += ( ( df3* )force_mat )[ j + count * tid ];
			}
		}
		( ( df3* )force_vec )[ tid ] = temp_force;
	}
}
__global__ void dev_calcLennardForceSorted( void *force_mat , void const *pos , SortCube const *sort_cube , int count )
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	if( bid < SortCube::DIV * SortCube::DIV * SortCube::DIV )
	{
		int indx[ 4 ];
		int rc = bid % ( SortCube::DIV * SortCube::DIV );
		indx[ 0 ] = rc % SortCube::DIV;
		indx[ 1 ] = rc / SortCube::DIV;
		indx[ 2 ] = bid / ( SortCube::DIV * SortCube::DIV );
		int const *main_arr = sort_cube->_array[ indx[ 0 ] ][ indx[ 1 ] ][ indx[ 2 ] ];
		indx[ 3 ] = main_arr[ 0 ];
		if( tid >= indx[ 3 ] )
		{
			return;
		}
		int main_index = main_arr[ tid + 1 ];
		df3 curpos = ( ( df3* )pos )[ main_index ];
		df3 final_force( 0.0f );
		for( int dx = -1; dx < 2; dx++ )
		{
			for( int dy = -1; dy < 2; dy++ )
			{
				for( int dz = -1; dz < 2; dz++ )
				{
					if( !(
						indx[ 0 ] + dx < 0 || indx[ 0 ] + dx > SortCube::DIV
						|| indx[ 1 ] + dy < 0 || indx[ 1 ] + dy > SortCube::DIV
						|| indx[ 2 ] + dz < 0 || indx[ 2 ] + dz > SortCube::DIV
						)
						)
					{
						int const *arr = sort_cube->_array[ indx[ 0 ] + dx ][ indx[ 1 ] + dy ][ indx[ 2 ] + dz ];
						int n = arr[ 0 ];
						for( int i = 0; i < n; ++i )
						{
							int indx = arr[ i + 1 ];
							if( indx != main_index )
							{
								df3 local_pos = ( ( df3* )pos )[ indx ];
								final_force += calcForce( curpos , local_pos );
								//( ( df3* )force_mat )[ indx * count + main_index ] = -force;
								//( ( df3* )force_mat )[ main_index * count + indx ] = force;
							}
						}
					}
				}
			}
			( ( df3* )force_mat )[ main_index ] = final_force;
		}
		/*__shared__ int indx[ 4 ];
		__shared__ int const *main_arr;
		__shared__ df3 neib[ 32 ][ 32 ];
		__shared__ int neibindx[ 32 ][ 32 ];
		__shared__ int neibn[ 32 ];
		__shared__ int tempdr[ 32 ][ 3 ];
		df3 curpos;
		int main_index;
		if( tid == 0 )
		{
			int rc = bid % ( SortCube::DIV * SortCube::DIV );
			indx[ 0 ] = rc % SortCube::DIV;
			indx[ 1 ] = rc / SortCube::DIV;
			indx[ 2 ] = bid / ( SortCube::DIV * SortCube::DIV );
			main_arr = sort_cube->_array[ indx[ 0 ] ][ indx[ 1 ] ][ indx[ 2 ] ];
			indx[ 3 ] = main_arr[ 0 ];
			int t = 0;
			for( int dx = -1; dx < 2; dx++ )
			{
				for( int dy = -1; dy < 2; dy++ )
				{
					for( int dz = -1; dz < 2; dz++ )
					{
						tempdr[ t ][ 0 ] = dx;
						tempdr[ t ][ 1 ] = dy;
						tempdr[ t ][ 2 ] = dz;
						t++;
					}
				}
			}
		}
		//__syncthreads();
		neibn[ tid ] = 0;
		if( tid < 27 &&
			!(
			indx[ 0 ] + tempdr[ tid ][ 0 ] < 0 || indx[ 0 ] + tempdr[ tid ][ 0 ] > SortCube::DIV
			|| indx[ 1 ] + tempdr[ tid ][ 1 ] < 0 || indx[ 1 ] + tempdr[ tid ][ 1 ] > SortCube::DIV
			|| indx[ 2 ] + tempdr[ tid ][ 2 ] < 0 || indx[ 2 ] + tempdr[ tid ][ 2 ] > SortCube::DIV
			)
			)
		{
			int const *arr = sort_cube->_array[ indx[ 0 ] + tempdr[ tid ][ 0 ] ][ indx[ 1 ] + tempdr[ tid ][ 1 ] ][ indx[ 2 ] + tempdr[ tid ][ 2 ] ];
			int n = arr[ 0 ];
			neibn[ tid ] = n;
			for( int i = 0; i < n; ++i )
			{
				int indx = arr[ i + 1 ];
				neib[ tid ][ i ] = ( ( df3* )pos )[ indx ];
				neibindx[ tid ][ i ] = indx;
			}
		}
		__syncthreads();
		if( tid >= indx[ 3 ] )
		{
			return;
		}
		main_index = main_arr[ tid + 1 ];
		curpos = ( ( df3* )pos )[ main_index ];
		df3 final_force( 0.0f );

		for( int k = 0; k < 27; k++ )
		{
			int n = neibn[ k ];
			for( int i = 0; i < n; ++i )
			{
				if( neibindx[ k ][ i ] != main_index )
				{
					final_force += calcForce( curpos , neib[ k ][ i ] );
				}
				//df3 iterpos = ( ( df3* )pos )[ local_index ];
				//df3 force = calcForce( curpos , iterpos );
				//( ( df3* )force_mat )[ local_index * count + main_index ] = -force;
				//( ( df3* )force_mat )[ main_index * count + local_index ] = force;
			}
		}*/
		//( ( df3* )force_mat )[ main_index ] = final_force;
	}
}
int N = 0;
void *dev_buffer;
SortCube local_sort_cube;
void LennardForceInit( int n )
{
	::N = n;
	hipMalloc( &dev_buffer , sizeof( SortCube ) + ( n + n * n ) * sizeof( df3 ) );
}
void LennardForceRelease()
{
	hipFree( dev_buffer );
}
void calcLennardForceVector( void const *pos , void *out_force , int count )
{
	if( count > N )
	{
		LennardForceRelease();
		LennardForceInit( count );
	}
	hipEvent_t start , stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start );
	{
		memset( &local_sort_cube , 0 , sizeof( SortCube ) );
		for( int i = 0; i < count; ++i )
		{
			int indx[ 3 ];
			getIndex( indx , ( float const * )pos + i * 3 );
			if( local_sort_cube._array[ indx[ 0 ] ][ indx[ 1 ] ][ indx[ 2 ] ][ 0 ] < SortCube::N )
			{
				local_sort_cube._array[ indx[ 0 ] ][ indx[ 1 ] ][ indx[ 2 ] ][ local_sort_cube._array[ indx[ 0 ] ][ indx[ 1 ] ][ indx[ 2 ] ][ 0 ] + 1 ] = i;
				local_sort_cube._array[ indx[ 0 ] ][ indx[ 1 ] ][ indx[ 2 ] ][ 0 ]++;
			} else
			{
				std::cout << "SORTCUBE SPACE IS OVER!!\n";
			}
		}
		df3 *dev_pos = ( df3* )dev_buffer;
		df3 *dev_force_mat = ( df3* )dev_buffer + count;
		SortCube *dev_sort_cube = ( SortCube* )( ( df3* )dev_buffer + count + count * count );
		hipMemcpy( dev_pos , pos , count * sizeof( df3 ) , hipMemcpyHostToDevice );
		hipMemset( dev_force_mat , 0 , count * count * sizeof( df3 ) );
		hipMemcpy( dev_sort_cube , &local_sort_cube , sizeof( SortCube ) , hipMemcpyHostToDevice );
		int block_count = SortCube::DIV * SortCube::DIV * SortCube::DIV;
		dev_calcLennardForceSorted << < block_count , 32 >> >( dev_force_mat , dev_pos , dev_sort_cube , count );
		hipDeviceSynchronize();
		//hipMemcpy( out_force , ( df3* )dev_buffer + count , count * sizeof( df3 ) , hipMemcpyDeviceToHost );
		//dim3 block_size = dim3( count / 32 + 1 , count / 32 + 1 );
		//dev_calcLennardForceVector << < block_size.x , 32 >> >( dev_force_mat , dev_pos , count );
		//hipDeviceSynchronize();
		hipMemcpy( out_force , dev_force_mat , count * sizeof( df3 ) , hipMemcpyDeviceToHost );
		/*float *force = ( float* )out_force;
		ito( 100 )
		{
			int k = i + 100;
			std::cout << force[ k * 3 ] << "," << force[ k * 3 + 1 ] << "," << force[ k * 3 + 2 ] << "\n";
		}*/

		/*hipMemcpy( dev_buffer , pos , count * sizeof( df3 ) , hipMemcpyHostToDevice );
		dim3 block_size = dim3( count / 32 + 1 , count / 32 + 1 );
		dev_calcLennardForceMatrix << < block_size , dim3( 32 , 32 ) >> >( dev_buffer , ( df3* )dev_buffer + count , count );
		hipDeviceSynchronize();
		dev_calcLennardForceVector << < block_size.x , 32 >> >( ( df3* )dev_buffer + count , dev_buffer , count );
		hipDeviceSynchronize();
		hipMemcpy( out_force , dev_buffer , count * sizeof( df3 ) , hipMemcpyDeviceToHost );*/
	}
	hipEventRecord( stop );
	hipEventSynchronize( stop );
	float milliseconds = 0;
	hipEventElapsedTime( &milliseconds , start , stop );
	std::cout << milliseconds << "\n";
}