#include "hip/hip_runtime.h"
#include <cudaRayTracerScene\scene.h>
#include <cuda_occupancy.h>
/*DEVICE void Scene::addObj( Object *obj )
{
	this->_obj_list.push_back( obj );
}*/
/*DEVICE f3 Scene::traceRay( Ray const &ray , int depth , unsigned int rand_seed , unsigned int SAMPLES_COUNT ) const
{
	Collision col;
	Material mat;
	{
		Collision tcol;
		bool success = false;
		float td2 = 9999999999.0f;
		for( int i = 0; i < _obj_count; i++ )
		{
			if( _obj_list[ i ]->getCollision( ray , tcol ) )
			{
				success = true;
				if( tcol._dist2 < td2 )
				{
					td2 = tcol._dist2;
					mat = _obj_list[ i ]->_material;
					col = tcol;
				}
			}
		}
		if( !success )
		{
			return f3( 0.2f , 0.23f , 0.46f ) + 0.6f * ray._v.z() * f3( 1.9f , 1.6f , 1.5f );
		}
	}
	if( mat._emit )
		return mat._color;
	f3 out( 0.0f );
	if( depth > 0 )
	{
		Ray temp_ray;
		temp_ray._pos = col._pos;
		f3 refl( 0.0f );
		if( mat._transperency < 1.0f )
		{
			for( int i = 0; i < SAMPLES_COUNT; i++ )
			{
				f3 v = VectorFactory::getDiffuseReflected( ray._v , col._n , mat._spec , rand_seed );
				temp_ray._v = v;
				refl += traceRay( temp_ray , depth - 1 , rand_seed , max( 1 , SAMPLES_COUNT >> 1 ) );
			}
		}
		f3 refr( 0.0f );
		if( mat._transperency > 0.0f )
		{
			float kn;
			if( !col._inside )
				kn = 1.0f / mat._n;
			else
				kn = mat._n;
			for( int i = 0; i < SAMPLES_COUNT; i++ )
			{
				f3 v = VectorFactory::getDiffuseRefracted( ray._v , col._n , mat._spec , kn , rand_seed );
				temp_ray._v = v;
				refr += traceRay( temp_ray , depth - 1 , rand_seed , max( 1 , SAMPLES_COUNT >> 1 ) );
			}
		}
		out = mat._color & ( refr * mat._transperency + refl * ( 1.0f - mat._transperency ) );
		out.mul( 1.0f / SAMPLES_COUNT );
	}
	return out;
}*/
DEVICE void Scene::traceRayQueued( QueueRay const &qray , QueueRay *rqueue , QueuePixel * pqueue , int index , int SAMPLES_COUNT ) const
{
	Collision col;
	Material mat;
	{
		Collision tcol;
		bool success = false;
		float td2 = 9999999999.0f;
		for( int i = 0; i < _obj_count; i++ )
		{
			if( _obj_list[ i ]->getCollision( { qray._pos , qray._v } , tcol ) )
			{
				success = true;
				if( tcol._dist2 < td2 )
				{
					td2 = tcol._dist2;
					mat = _obj_list[ i ]->_material;
					col = tcol;
				}
			}
		}
		if( !success )
		{
			pqueue[ index ] = { qray._target_pixel , qray._color_k & ( f3( 0.2f , 0.23f , 0.46f ) + qray._v.z() * f3( 1.9f , 1.6f , 1.5f ) ) };
			//rqueue->writePixel( qray._color_k & ( f3( 0.2f , 0.23f , 0.46f ) + qray._v.z() * f3( 1.9f , 1.6f , 1.5f ) ) , qray._dir_pixel );
			return;
		}
		if( mat._emit )
		{
			pqueue[ index ] = { qray._target_pixel , qray._color_k & mat._color };
			//rqueue->writePixel( qray._color_k & mat._color , qray._dir_pixel );
			return;
		}
	}
	f3 out( 0.0f );
	{
		QueueRay temp_ray;
		temp_ray._pos = col._pos;
		temp_ray._target_pixel = qray._target_pixel;
		temp_ray._have_value = 1;
		f3 refl( 0.0f );
		if( mat._transperency < 1.0f )
		{
			temp_ray._color_k = qray._color_k & ( mat._color * ( 1.0f - mat._transperency ) );
			for( int i = 0; i < SAMPLES_COUNT; i++ )
			{
				f3 v = VectorFactory::getDiffuseReflected( qray._v , col._n , mat._spec , index );
				temp_ray._v = v;
				rqueue[ index * SAMPLES_COUNT + i ] = temp_ray;
			}
		}
		f3 refr( 0.0f );
		if( mat._transperency > 0.0f )
		{
			temp_ray._color_k = qray._color_k & ( mat._color * mat._transperency );
			float kn;
			if( !col._inside )
				kn = 1.0f / mat._n;
			else
				kn = mat._n;
			for( int i = 0; i < SAMPLES_COUNT; i++ )
			{
				f3 v = VectorFactory::getDiffuseRefracted( qray._v , col._n , mat._spec , kn , index );
				temp_ray._v = v;
				rqueue[ index * SAMPLES_COUNT + SAMPLES_COUNT + i ] = temp_ray;
			}
		}
	}
}
DEVICE bool Sphere::getCollision( Ray const &ray , Collision &out ) const
{
	const float EPS = 0.001f;
	f3 dp = ray._pos - _pos;
	float b = 2.0f * dp * ray._v;
	float c = dp * dp - _radius * _radius;
	float d = b * b - 4.0f * c;
	if( d < 0.0f )
		return false;
	d = sqrtf( d );
	float t = ( -b - d ) * 0.5f;
	if( t < EPS )
		t += d;
	if( t < EPS )
		return false;
	out._dist2 = t * t;
	out._pos = ray._pos + ray._v * t;
	out._n = ( out._pos - _pos ).g_norm();
	if( c < EPS )
	{
		out._n.mul( -1.0f );
		out._inside = true;
	}
	out._inside = false;
	return true;
}
DEVICE bool InfPlane::getCollision( Ray const &ray , Collision &out ) const
{
	float d = ray._v * _n;
	f3 dp = _pos - ray._pos;
	float p = dp * _n;
	float t = p / d;
	if( t < 0.0001f )
		return false;
	out._pos = ray._pos + ray._v * t;
	out._dist2 = t * t;
	if( p < 0.0f )
		out._n = _n;
	else
		out._n = -_n;
	out._inside = false;
	return true;
}