#include "hip/hip_runtime.h"
#include <cudaRayTracerScene\scene.h>
/*DEVICE void Scene::addObj( Object *obj )
{
	this->_obj_list.push_back( obj );
}*/
DEVICE f4 Scene::traceRay( f3 const &pos , f3 const &v )
{

}
DEVICE Collision Sphere::getCollision( Ray const &ray ) const
{
	Collision out{ false };
	f3 dp = ray._pos - _pos;
	float b = 2.0f * dp * ray._v;
	if( b > 2.0f * _radius )
		return out;
	float c = dp * dp + _radius * _radius;
	float d = b * b - 4.0f * c;
	if( d < 0.0f )
		return out;
	d = sqrtf( d );
	float t = ( - b - d ) * 0.5f;
	if( t < 0.0f )
		t += d;
	out._dist2 = t * t;
	out._pos = ray._pos + ray._v * t;
	out._n = ( out._pos - _pos ).g_norm();
	out._collided = true;
	return out;
}