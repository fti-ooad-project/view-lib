#include "hip/hip_runtime.h"
#include <cudaRayTracerScene\scene.h>
/*DEVICE void Scene::addObj( Object *obj )
{
	this->_obj_list.push_back( obj );
}*/
DEVICE f4 Scene::traceRay( Ray const &ray , int depth , unsigned int rand_seed ) const
{
	if( depth < 0 )
		return f4( 0.0f );
	Collision col;
	{
		Collision tcol;
		bool success = false;
		float td2 = 9999999999.0f;
		for( int i = 0; i < _obj_count; i++ )
		{
			if( _obj_list[ i ]->getCollision( ray , tcol ) )
			{
				success = true;
				if( tcol._dist2 < td2 )
				{
					td2 = tcol._dist2;
					col = tcol;
				}
			}
		}
		if( !success )
		{
			return f4( 0.4f + 0.6f * abs( ray._v.z() ) );
		}
	}
	f4 out( 0.0f );
	{
		const unsigned int SAMPLES_COUNT = 100;
		for( int i = 0; i < SAMPLES_COUNT; i++ )
		{
			f3 rand_vec = VectorFactory::getRandomSphere( rand_seed );
			if( rand_vec * col._n < 0.0f ) rand_vec.mul( -1.0f );
			out += traceRay( { col._pos , rand_vec } , depth - 1 , rand_seed );
		}
		out.mul( 1.0f / SAMPLES_COUNT );
	}
	return out;
}
DEVICE bool Sphere::getCollision( Ray const &ray , Collision &out ) const
{
	f3 dp = ray._pos - _pos;
	float b = 2.0f * dp * ray._v;
	if( b > 2.0f * _radius - 0.01 )
		return false;
	float c = dp * dp - _radius * _radius;
	float d = b * b - 4.0f * c;
	if( d < 0.0f )
		return false;
	d = sqrtf( d );
	float t = ( - b - d ) * 0.5f;
	if( t < 0.0f )
		t += d;
	if( t < 0.001f )
		return false;
	out._dist2 = t * t;
	out._pos = ray._pos + ray._v * t;
	out._n = ( out._pos - _pos ).g_norm();
	out._collided = true;
	return true;
}