#include "hip/hip_runtime.h"
#include <cudaRayTracerScene\scene.h>
#include <cuda_occupancy.h>
/*DEVICE void Scene::addObj( Object *obj )
{
	this->_obj_list.push_back( obj );
}*/
/*DEVICE f3 Scene::traceRay( Ray const &ray , int depth , unsigned int rand_seed , unsigned int SAMPLES_COUNT ) const
{
	Collision col;
	Material mat;
	{
		Collision tcol;
		bool success = false;
		float td2 = 9999999999.0f;
		for( int i = 0; i < _obj_count; i++ )
		{
			if( _obj_list[ i ]->getCollision( ray , tcol ) )
			{
				success = true;
				if( tcol._dist2 < td2 )
				{
					td2 = tcol._dist2;
					mat = _obj_list[ i ]->_material;
					col = tcol;
				}
			}
		}
		if( !success )
		{
			return f3( 0.2f , 0.23f , 0.46f ) + 0.6f * ray._v.z() * f3( 1.9f , 1.6f , 1.5f );
		}
	}
	if( mat._emit )
		return mat._color;
	f3 out( 0.0f );
	if( depth > 0 )
	{
		Ray temp_ray;
		temp_ray._pos = col._pos;
		f3 refl( 0.0f );
		if( mat._transperency < 1.0f )
		{
			for( int i = 0; i < SAMPLES_COUNT; i++ )
			{
				f3 v = VectorFactory::getDiffuseReflected( ray._v , col._n , mat._spec , rand_seed );
				temp_ray._v = v;
				refl += traceRay( temp_ray , depth - 1 , rand_seed , max( 1 , SAMPLES_COUNT >> 1 ) );
			}
		}
		f3 refr( 0.0f );
		if( mat._transperency > 0.0f )
		{
			float kn;
			if( !col._inside )
				kn = 1.0f / mat._n;
			else
				kn = mat._n;
			for( int i = 0; i < SAMPLES_COUNT; i++ )
			{
				f3 v = VectorFactory::getDiffuseRefracted( ray._v , col._n , mat._spec , kn , rand_seed );
				temp_ray._v = v;
				refr += traceRay( temp_ray , depth - 1 , rand_seed , max( 1 , SAMPLES_COUNT >> 1 ) );
			}
		}
		out = mat._color & ( refr * mat._transperency + refl * ( 1.0f - mat._transperency ) );
		out.mul( 1.0f / SAMPLES_COUNT );
	}
	return out;
}*/
DEVICE void Scene::traceRayQueued( QueueRay const &qray , RayQueue *rqueue , unsigned int rand_seed , unsigned int SAMPLES_COUNT ) const
{
	Collision col;
	Material mat;
	{
		Collision tcol;
		bool success = false;
		float td2 = 9999999999.0f;
		for( int i = 0; i < _obj_count; i++ )
		{
			if( _obj_list[ i ]->getCollision( { qray._pos , qray._v } , tcol ) )
			{
				success = true;
				if( tcol._dist2 < td2 )
				{
					td2 = tcol._dist2;
					mat = _obj_list[ i ]->_material;
					col = tcol;
				}
			}
		}
		if( !success )
		{
			rqueue->writePixel( qray._color_k & ( f3( 0.2f , 0.23f , 0.46f ) + qray._v.z() * f3( 1.9f , 1.6f , 1.5f ) ) , qray._dir_pixel );
			return;
		}
		if( mat._emit )
		{
			rqueue->writePixel( qray._color_k & mat._color , qray._dir_pixel );
			return;
		}
	}
	f3 out( 0.0f );
	{
		QueueRay temp_ray;
		temp_ray._pos = col._pos;
		temp_ray._dir_pixel = qray._dir_pixel;
		f3 refl( 0.0f );
		if( mat._transperency < 1.0f )
		{
			temp_ray._color_k = qray._color_k & ( mat._color * ( 1.0f - mat._transperency ) );
			for( int i = 0; i < SAMPLES_COUNT; i++ )
			{
				f3 v = VectorFactory::getDiffuseReflected( qray._v , col._n , mat._spec , rand_seed );
				temp_ray._v = v;
				rqueue->add( temp_ray );
			}
		}
		f3 refr( 0.0f );
		if( mat._transperency > 0.0f )
		{
			temp_ray._color_k = qray._color_k & ( mat._color * mat._transperency );
			float kn;
			if( !col._inside )
				kn = 1.0f / mat._n;
			else
				kn = mat._n;
			for( int i = 0; i < SAMPLES_COUNT; i++ )
			{
				f3 v = VectorFactory::getDiffuseRefracted( qray._v , col._n , mat._spec , kn , rand_seed );
				temp_ray._v = v;
				rqueue->add( temp_ray );
			}
		}
	}
}
DEVICE bool Sphere::getCollision( Ray const &ray , Collision &out ) const
{
	const float EPS = 0.001f;
	f3 dp = ray._pos - _pos;
	float b = 2.0f * dp * ray._v;
	float c = dp * dp - _radius * _radius;
	float d = b * b - 4.0f * c;
	if( d < 0.0f )
		return false;
	d = sqrtf( d );
	float t = ( -b - d ) * 0.5f;
	if( t < EPS )
		t += d;
	if( t < EPS )
		return false;
	out._dist2 = t * t;
	out._pos = ray._pos + ray._v * t;
	out._n = ( out._pos - _pos ).g_norm();
	if( c < EPS )
	{
		out._n.mul( -1.0f );
		out._inside = true;
	}
	out._inside = false;
	return true;
}
DEVICE bool InfPlane::getCollision( Ray const &ray , Collision &out ) const
{
	float d = ray._v * _n;
	f3 dp = _pos - ray._pos;
	float p = dp * _n;
	float t = p / d;
	if( t < 0.0001f )
		return false;
	out._pos = ray._pos + ray._v * t;
	out._dist2 = t * t;
	if( p < 0.0f )
		out._n = _n;
	else
		out._n = -_n;
	out._inside = false;
	return true;
}
DEVICE void RayQueue::add( QueueRay const &qray )
{
	if( _pos < _array_size )
	{
		int i = atomicAdd( &_pos , 1 );
		if( i < _array_size )
		{
			_array[ i ] = qray;
		} else
		{
			atomicAdd( &_pos , -1 );
		}
	}
}
DEVICE unsigned int RayQueue::getSize() const
{
	return _pos;
}
DEVICE void RayQueue::writePixel( f3 const &pix , unsigned int indx )
{
	atomicAdd( ( float* )( _screen + indx ) , pix.x() );
	atomicAdd( ( float* )( _screen + indx  ) + 1 , pix.y() );
	atomicAdd( ( float* )( _screen + indx ) + 2 , pix.z() );
	atomicAdd( ( float* )( _screen + indx  ) + 3 , 1.0f );
}
DEVICE void RayQueue::empty()
{
	_pos = 0;
}